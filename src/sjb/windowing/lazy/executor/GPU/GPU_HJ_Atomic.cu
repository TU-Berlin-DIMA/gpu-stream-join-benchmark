#include <chrono>
#include <algorithm>
#include <sjb/windowing/lazy/executor/GPU/GPU_HJ_Atomic.cuh>
#include <sjb/sink/Sink.h>
#include <sjb/utils/ErrorChecking.cuh>
#include <sjb/utils/Logger.hpp>
#include <cassert>

#define xstr(a) str(a)
#define str(a) #a

namespace Windowing {
    namespace Lazy {
        namespace Executor {
            GPU_HJ_Atomic::GPU_HJ_Atomic(
                    uint64_t numDistinctKeys,
                    uint64_t batchSize,
                    uint64_t maxTupleInWindow) : GPU_HJ(numDistinctKeys, batchSize, maxTupleInWindow) {

            }

            ExecutionStatistic GPU_HJ_Atomic::execute(Tuple *leftRingBuffer, Tuple *rightRingBuffer,
                                                        std::vector<uint64_t> leftIndexesToJoin,
                                                        std::vector<uint64_t> rightIndexesToJoin,
                                                        Sink &sink) {
                auto nLeftTuples = leftIndexesToJoin.size() * batchSize;
                auto nRightTuples = rightIndexesToJoin.size() * batchSize;

                // Assert that we have enough hash table
                LOG_DEBUG("maxTupleInWindow: %lu, leftIndexesToJoin.size(): %lu, batchSize:%lu", maxTupleInWindow,
                          leftIndexesToJoin.size(), batchSize);
                assert(maxTupleInWindow >= nLeftTuples);

                // Start the timer
                auto t0 = std::chrono::high_resolution_clock::now();

                // Copy the build side to the local store of the joiner
                copyTuplesToLocalStore(d_leftTuples, leftRingBuffer, leftIndexesToJoin, batchSize, leftStream);
                copyTuplesToLocalStore(d_rightTuples, rightRingBuffer, rightIndexesToJoin, batchSize, rightStream);

                CUDA_CHECK(hipStreamSynchronize(leftStream));

                // Reset the histogram
                CUDA_CHECK(hipMemsetAsync(d_histogram, 0, numDistinctKeys * sizeof(unsigned long long), joinerStream));
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                launchComputeHistogramKernel(d_leftTuples, nLeftTuples, d_histogram, numDistinctKeys, joinerStream);
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                // Copy the computed histogram back to host
                CUDA_CHECK(hipMemcpyAsync(h_histogram, d_histogram, numDistinctKeys * sizeof(unsigned  long long), hipMemcpyDeviceToHost, joinerStream));
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                // Compute prefix sum from the histogram
                memset(h_prefixSum, 0, numDistinctKeys * sizeof(uint64_t));
                uint64_t runningSum = 0;
                for (uint64_t kIdx = 0; kIdx < numDistinctKeys; kIdx++) {
                    h_prefixSum[kIdx] = runningSum;
                    runningSum += h_histogram[kIdx];
                }

                // Copy the prefixSum to the GPU
                CUDA_CHECK(hipMemcpyAsync(d_prefixSum, h_prefixSum, numDistinctKeys * sizeof(uint64_t), hipMemcpyHostToDevice, joinerStream));
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                // Reset the occupation manager
                CUDA_CHECK(hipMemsetAsync(d_occupation, 0, numDistinctKeys * sizeof(unsigned long long), joinerStream));
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                // Build the hashtable
                launchBuildHashTableKernel(d_leftTuples, nLeftTuples, d_hashTable,
                                           d_prefixSum, d_occupation, numDistinctKeys, maxTupleInWindow,
                                           joinerStream);
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                // Reset the result counter to 0
                CUDA_CHECK(hipMemsetAsync(d_resultCount, 0, sizeof(unsigned long long), joinerStream));
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                // End of build time
                auto t1 = std::chrono::high_resolution_clock::now();

                // Make sure that the probe-side tuple has arrived to the GPU
                CUDA_CHECK(hipStreamSynchronize(rightStream));


                // Launch the probe count kernel
                launchProbeWriteKernel(d_rightTuples, nRightTuples, nLeftTuples,
                                       d_hashTable, d_histogram, d_prefixSum,
                                       sink.counterGPU, numDistinctKeys, joinerStream,
                                       sink.sinkBuffer, sink.sinkBufferSize);
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                // End of probe time
                auto t2 = std::chrono::high_resolution_clock::now();

                // Write the result count to experiment statistics
                ExecutionStatistic es = ExecutionStatistic();
                es.resultCount = *sink.counterGPU;

                LOG_DEBUG("Build time: %ld ms", std::chrono::duration_cast<std::chrono::milliseconds>(t1 - t0).count());
                LOG_DEBUG("Probe time: %ld ms", std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count());
                LOG_DEBUG("Counter: %llu", *h_resultCount);

                auto buildTime = std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
                auto probeTime = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();

                totalBuildTime += buildTime;
                totalProbeTime += probeTime;
                totalExecutedWindows++;

                es.buildTime = buildTime;
                es.probeTime = probeTime;
                return es;
            }

            GPU_HJ_Atomic::~GPU_HJ_Atomic() = default;
        }
    }
}