#include "hip/hip_runtime.h"
#include <atomic>
#include <chrono>

#include <sjb/windowing/lazy/executor/GPU/GPU_SMJ_EstimatedSelectivity.cuh>
#include <sjb/utils/ErrorChecking.cuh>
#include <sjb/utils/Logger.hpp>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <sjb/utils/ResultTuple.hpp>
#include <sjb/sink/Sink.h>
#include <sjb/utils/CudaUtils.cuh>

#define xstr(a) str(a)
#define str(a) #a

namespace Windowing {
    namespace Lazy {
        namespace Executor {
            GPU_SMJ_EstimatedSelectivity::GPU_SMJ_EstimatedSelectivity(uint64_t numDistinctKeys, uint64_t batchSize,
                                                                       uint64_t maxTupleInWindow,
                                                                       double estimatedSelectivity) :
                    GPU_SMJ(numDistinctKeys, batchSize, maxTupleInWindow),
                    estimatedSelectivity(estimatedSelectivity) {
            }

            GPU_SMJ_EstimatedSelectivity::~GPU_SMJ_EstimatedSelectivity() {
            }


            ExecutionStatistic
            GPU_SMJ_EstimatedSelectivity::execute(Tuple *leftRingBuffer, Tuple *rightRingBuffer,
                                                  std::vector<uint64_t> leftIndexesToCopy,
                                                  std::vector<uint64_t> rightIndexesToCopy, Sink &sink) {
                auto t0 = std::chrono::high_resolution_clock::now();

                auto nLeftTuples = leftIndexesToCopy.size() * batchSize;
                auto nRightTuples = rightIndexesToCopy.size() * batchSize;

                CUDA_CHECK(
                        hipMemsetAsync(d_leftOccupation, 0, numDistinctKeys * sizeof(unsigned long long), leftStream));
                CUDA_CHECK(hipMemsetAsync(d_rightOccupation, 0, numDistinctKeys * sizeof(unsigned long long),
                                           rightStream));

                CUDA_CHECK(hipMemsetAsync(d_leftHistogram, 0, numDistinctKeys * sizeof(uint64_t), leftStream));
                CUDA_CHECK(hipMemsetAsync(d_rightHistogram, 0, numDistinctKeys * sizeof(uint64_t), rightStream));

                // Copy data from ring buffer to the GPU
                copyTuplesToLocalStore(d_leftTuples, leftRingBuffer, leftIndexesToCopy, batchSize, leftStream);
                CUDA_CHECK(hipStreamSynchronize(leftStream));

                launchHistogramKernel(d_leftTuples, nLeftTuples, d_leftHistogram,leftStream);
                CUDA_CHECK(hipStreamSynchronize(leftStream));


                auto leftPolicy = thrust::cuda::par.on(leftStream);
                thrust::exclusive_scan(leftPolicy, d_leftHistogram, d_leftHistogram + numDistinctKeys, d_leftPrefixSum);

                launchSortKernel(d_leftTuples,
                                 d_sortedLeftTuples,
                                 d_leftPrefixSum,
                                 nLeftTuples,
                                 d_leftOccupation,
                                 leftStream);
                CUDA_CHECK(hipStreamSynchronize(leftStream));

                copyTuplesToLocalStore(d_rightTuples, rightRingBuffer, rightIndexesToCopy, batchSize, rightStream);
                CUDA_CHECK(hipStreamSynchronize(rightStream));

                launchHistogramKernel(d_rightTuples, nRightTuples, d_rightHistogram, rightStream);
                CUDA_CHECK(hipStreamSynchronize(rightStream));

                auto rightPolicy = thrust::cuda::par.on(rightStream);
                thrust::exclusive_scan(rightPolicy, d_rightHistogram, d_rightHistogram + numDistinctKeys,
                                       d_rightPrefixSum);

                launchSortKernel(d_rightTuples, d_sortedRightTuples, d_rightPrefixSum, nRightTuples, d_rightOccupation, rightStream);
                CUDA_CHECK(hipStreamSynchronize(rightStream));


                CUDA_CHECK(hipStreamSynchronize(leftStream));
                CUDA_CHECK(hipStreamSynchronize(rightStream));

                auto t1 = std::chrono::high_resolution_clock::now();



                // Allocate output buffer of a size of maximum possible matches
                // Allocate twice the estimated result to allow for minor error in the estimation
                ResultTuple *d_resultTuple;
                uint64_t expectedResultCount = 1.125 * ceil(estimatedSelectivity * nLeftTuples * nRightTuples);
                LOG_DEBUG("Expected result count: %lu", expectedResultCount);
                CUDA_CHECK(hipMalloc(&d_resultTuple, expectedResultCount * sizeof(ResultTuple)));

                // Reset the result counter to 0 before joining
                CUDA_CHECK(hipMemsetAsync(d_resultCount, 0, sizeof(unsigned long long), joinerStream));
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                launchMergeJoinKernel(d_leftHistogram, d_rightHistogram, d_sortedLeftTuples,
                                      d_sortedRightTuples, d_resultCount, numDistinctKeys, d_resultTuple, joinerStream);
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                CUDA_CHECK(hipMemcpyAsync(h_resultCount, d_resultCount, sizeof(unsigned long long),
                                           hipMemcpyDeviceToHost,
                                           joinerStream));
                CUDA_CHECK(hipStreamSynchronize(joinerStream));

                uint64_t nWrittenTuple = 0;
                uint64_t copyCount;
                while (nWrittenTuple < *h_resultCount) {
                    if (nWrittenTuple + sink.getSinkBufferSize() > *h_resultCount) {
                        copyCount = *h_resultCount-nWrittenTuple;
                    } else {
                        copyCount = sink.getSinkBufferSize();
                    }
                    sink.incrementCounterAndStore(d_resultTuple+nWrittenTuple, copyCount);
                    nWrittenTuple += copyCount;
                }
                assert(nWrittenTuple == *h_resultCount);

                auto t2 = std::chrono::high_resolution_clock::now();

                totalSortTime += std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();
                totalMergeTime += std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
                totalExecutedWindows++;

                ExecutionStatistic es = ExecutionStatistic();
                es.sortTime = std::chrono::duration_cast<std::chrono::microseconds>(t1 - t0).count();;
                es.mergeTime = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();;

                hipFree(d_resultTuple);
                return es;
            }
        }
    }
}
